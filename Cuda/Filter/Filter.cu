#include "hip/hip_runtime.h"


// CUDA runtime
#include <hip/hip_runtime.h>

// Utilities and system includes
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "Constants.h"
#include "Filter.h"

__constant__ double dev_kernel[KERNEL_LENGTH*KERNEL_LENGTH];

extern "C" void setConvolutionKernel(double* h_Kernel)
{
    //for (int i = 0; i < KERNEL_LENGTH; i++) {
    	hipMemcpyToSymbol(HIP_SYMBOL(dev_kernel), h_Kernel, KERNEL_LENGTH*KERNEL_LENGTH*sizeof(double));
	//}
}

extern "C" void setConvolutionKernel2(double h_Kernel[KERNEL_LENGTH*KERNEL_LENGTH])
{
    //for (int i = 0; i < KERNEL_LENGTH; i++) {
    	hipMemcpyToSymbol(HIP_SYMBOL(dev_kernel), h_Kernel, KERNEL_LENGTH*KERNEL_LENGTH*sizeof(double));
	//}
}

__global__ void runConvolutionGPU(double* image, double* result, int height, int width, int step)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	//int O_TILE_WIDTH = blockDim.x-(lkernel/2)*2;
	//int O_TILE_HEIGHT = blockDim.y-(lkernel/2)*2;
	int row_o = threadIdx.y + blockIdx.y*O_TILE_HEIGHT;
	int col_o = threadIdx.x + blockIdx.x*O_TILE_WIDTH;

	int row_i = row_o - KERNEL_LENGTH/2;
	int col_i = col_o - KERNEL_LENGTH/2;

	__shared__ double N_ds[BLOCK_DIM_Y][BLOCK_DIM_X];

	if((row_i >= 0) && (row_i < height) && (col_i >= 0) && (col_i < height)){
		N_ds[ty][tx] = image[row_i*width+col_i];
	}else{
		N_ds[ty][tx] = 0.0f;
	}

	__syncthreads();

	double output = 0.0f;
	if(ty < O_TILE_HEIGHT && tx < O_TILE_WIDTH){
		for(int i=0; i<KERNEL_LENGTH; i++){
			for(int j=0; j<KERNEL_LENGTH; j++){
				output += dev_kernel[i*KERNEL_LENGTH+j]*N_ds[(i+ty)][(j+tx)];
			}
		}
		if(row_o < height && col_o < width){
			result[row_o*width+col_o] = output;
		}
	}
}

extern "C" void convolutionGPU(double* image, double* result, int x_length, int y_length, int step)
{
	double* dev_image, *dev_result;

	hipMalloc((void**)&dev_image, x_length*y_length*sizeof(double));
	hipMalloc((void**)&dev_result, x_length*y_length*sizeof(double));

	hipMemcpy(dev_image, image, x_length*y_length*sizeof(double), hipMemcpyHostToDevice);

	dim3 blocks(y_length/O_TILE_HEIGHT + (((y_length%O_TILE_HEIGHT)==0)?0:1), x_length/O_TILE_HEIGHT + (((y_length%O_TILE_HEIGHT)==0)?0:1));
	dim3 threads(BLOCK_DIM_Y,BLOCK_DIM_X);
	runConvolutionGPU<<<blocks,threads>>>(dev_image, dev_result, y_length, x_length, step);

	hipMemcpy(result, dev_result, x_length*y_length*sizeof(double), hipMemcpyDeviceToHost);

	hipFree(dev_image);
	hipFree(dev_result);
}
