#include "hip/hip_runtime.h"
#include "SaliencyMap.h"
#include <hip/hip_runtime.h>

void gpuHostAlloc(double*& d_p, int rows, int cols) {
	//double* dPointer;
	hipHostAlloc( (void**)&d_p, rows*cols*sizeof(double), hipHostMallocMapped );
	//return dPointer;
}

void gpuMalloc(double*& d_p, int rows, int cols){
    hipMalloc((void**)&d_p, rows*cols*sizeof(double));
}

void gpuFreeHostAlloc(double*& d_p){
    hipHostFree(d_p);
}

void gpuFreeMalloc(double*& d_p){
    hipFree(d_p);
}


/*
 *  CPU Extra-functions
 *  ===================
 */
void getMap(double* &feature, double* &map, 
                        const double kernel[][5],
                        int rows, int cols) {
    uint c;
    float dNorm25 = 0.0f, dNorm26 = 0.0f; 
    float dNorm36 = 0.0f, dNorm37 = 0.0f; 
    float dNorm47 = 0.0f, dNorm48 = 0.0f; 

    // Allocate Host-Device
    double *dFeature;
    double *dPyLevel1, *dPyLevel2;
    double *dPyLevel3, *dPyLevel4;
    double *dPyLevel5, *dPyLevel6;
    double *dPyLevel7, *dPyLevel8;

    double *feat25, *feat36, *feat47;

    c =   4*  4; hipMalloc(&dPyLevel2, rows*cols/c*sizeof(double));
    c =   8*  8; hipMalloc(&dPyLevel3, rows*cols/c*sizeof(double));
    c =  16* 16; hipMalloc(&dPyLevel4, rows*cols/c*sizeof(double));
    c =  32* 32; hipMalloc(&dPyLevel5, rows*cols/c*sizeof(double));
    c =  64* 64; hipMalloc(&dPyLevel6, rows*cols/c*sizeof(double));
    c = 128*128; hipMalloc(&dPyLevel7, rows*cols/c*sizeof(double));
    c = 256*256; hipMalloc(&dPyLevel8, rows*cols/c*sizeof(double));

    c = 4*4; hipMalloc(&feat25, rows*cols/c*sizeof(double));
    c = 4*4; hipMalloc(&feat36, rows*cols/c*sizeof(double));
    c = 4*4; hipMalloc(&feat47, rows*cols/c*sizeof(double));

    // Handles
    Filter blur(kernel);

    // Generate pyramid
    blur.convolution( feature , rows    , cols    , dPyLevel1, 2, THREAD_COUNT);
    blur.convolution(dPyLevel1, rows/2  , cols/2  , dPyLevel2, 2, THREAD_COUNT);
    blur.convolution(dPyLevel2, rows/4  , cols/4  , dPyLevel3, 2, THREAD_COUNT);
    blur.convolution(dPyLevel3, rows/8  , cols/8  , dPyLevel4, 2, THREAD_COUNT);
    blur.convolution(dPyLevel4, rows/16 , cols/16 , dPyLevel5, 2, THREAD_COUNT);
    blur.convolution(dPyLevel5, rows/32 , cols/32 , dPyLevel6, 2, THREAD_COUNT);
    blur.convolution(dPyLevel6, rows/64 , cols/64 , dPyLevel7, 2, THREAD_COUNT);
    blur.convolution(dPyLevel7, rows/128, cols/128, dPyLevel8, 2, THREAD_COUNT);

    // Center-surround difference
    centerSurroundDiff(dPyLevel2, dPyLevel5, feat25, 2, 5, 2);
    centerSurroundDiff(dPyLevel2, dPyLevel6, feat26, 2, 6, 2);

    centerSurroundDiff(dPyLevel3, dPyLevel6, feat36, 3, 6, 2);
    centerSurroundDiff(dPyLevel3, dPyLevel7, feat37, 3, 7, 2);

    centerSurroundDiff(dPyLevel4, dPyLevel7, feat47, 4, 7, 2);
    centerSurroundDiff(dPyLevel4, dPyLevel8, feat48, 4, 8, 2);

    // Free pyramid
    hipFree(dPyLevel1); hipFree(dPyLevel2);
    hipFree(dPyLevel3); hipFree(dPyLevel4);
    hipFree(dPyLevel5); hipFree(dPyLevel6);
    hipFree(dPyLevel7); hipFree(dPyLevel8);

    // Normalizarion
    c = 4*4;
    nrmSumGPU(feat25,feat26,map,rows*cols/c);
    nrmSumGPU(feat36,feat37,map,rows*cols/c);
    nrmSumGPU(feat47,feat48,map,rows*cols/c);

    // Free proto-feature
    hipFree(feat25); hipFree(feat26);
    hipFree(feat36); hipFree(feat37);
    hipFree(feat47); hipFree(feat48);
}

void centerSurroundDiffGPU(double* &dSupLevel, double* &dLowLevel,
                           double* &dDifference, 
                           int sup, int low, int endl,
                           int rows, int cols){
    int supRow = rows / pow2(sup);
    int supCol = cols / pow2(sup);

    int lowRow = rows / pow2(low);
    int lowCol = cols / pow2(low);

    // Interpolation
    double* dLowLevelGrownUp;
    hipMalloc(&dLowLevelGrownUp, supRow*supCol*sizeof(double));
    Filter::growthMatrix(dLowLevel, lowRow, lowCol, 
        dLowLevelGrownUp, pow2(low - sup));

    if (sup != endl) {
        double* dRawDifference;
        hipMalloc(&dRawDifference, supRow*supCol*sizeof(double));

        absDifference<<<BlocksInGrid,threadsPerBlock>>>(dRawDifference, dSupLevel, 
                                     dLowLevelGrownUp, supRow*supCol);
        Filter::growthMatrix(dRawDifference, supRow, supCol, 
                dDifference, pow2(sup - endl));

        hipFree(dRawDifference);
    }
    else {
        absDifference<<<BlocksInGrid,threadsPerBlock>>>(dDifference, dSupLevel, 
                                     dLowLevelGrownUp, supRow*supCol);
    }

    // Liberar memoria
    hipFree(dLowLevelGrownUp);
}


void nrmSumGPU(double* &dProSupFeature, double* &dProInfFeature, 
               double* &dMap,
               int rows, int cols){
    //
    // Calculo de norma infinito
    // -------------------------

    // Separar memoria
    double *dInfNormProSupFeature, *dInfNormProInfFeature;
    double *dMaxProSupFeature , *dMaxProInfFeature;
    double *dMeanProSupFeature, *dMeanProInfFeature;

    hipMalloc(&dInfNormProSupFeature, sizeof(double)*BLOCKSIZE*BLOCKSIZE);
    hipMalloc(&dInfNormProInfFeature, sizeof(double)*BLOCKSIZE*BLOCKSIZE);

    hipMalloc(&dMaxProSupFeature, sizeof(double));
    hipMalloc(&dMaxProInfFeature, sizeof(double));

    hipMalloc(&dMeanProSupFeature, sizeof(double));
    hipMalloc(&dMeanProInfFeature, sizeof(double));

    dim3 dimBlock(BLOCKSIZE,BLOCKSIZE);
    dim3 dimGrid (rows/dimBlock.x,cols/dimBlock.y);	

    matInfinityNorm<<<dimGrid,dimBlock>>>(dProSupFeature,dInfNormProSupFeature,
                                          rows, cols, BLOCKSIZE);

    matInfinityNorm<<<dimGrid,dimBlock>>>(dProInfFeature,dInfNormProInfFeature,
                                          rows, cols, BLOCKSIZE);

    //
    // Dividir con scalar
    // ------------------
    divScalarMatrix<<<BlocksInGrid,threadsPerBlock>>>(dProSupFeature,dInfNormProSupFeature, rows*cols);
    divScalarMatrix<<<BlocksInGrid,threadsPerBlock>>>(dProInfFeature,dInfNormProInfFeature, rows*cols);
    
    //
    // Maximum
    // -------
    find_maximum(dProSupFeature, dMaxProSupFeature, rows*cols);
    find_maximum(dProSupFeature, dMaxProInfFeature, rows*cols);

    //
    // Mean
    // ----
    meanMatrix<<<BlocksInGrid,threadsPerBlock>>>(dProSupFeature, dMeanProSupFeature, rows*cols);
    meanMatrix<<<BlocksInGrid,threadsPerBlock>>>(dProInfFeature, dMeanProInfFeature, rows*cols);

    //
    // Apply
    // -----
    applyNormSum<<<BlocksInGrid,threadsPerBlock>>>(dMap,dProSupFeature,dMaxProSupFeature,dMeanProSupFeature,
                                                   dProInfFeature,dMaxProInfFeature,dMeanProInfFeature,
                                                   rows*cols);

    // Liberar memoria
    hipFree(dInfNormProSupFeature);
    hipFree(dInfNormProInfFeature);

    hipFree(dMaxProSupFeature);
    hipFree(dMaxProInfFeature);

    hipFree(dMeanProSupFeature);
    hipFree(dMeanProInfFeature);
}


void getSalency(double* &salency, 
                double* &Imap, double* &Omap, double* &Cmap,
                int rows, int cols) {
	sum3<<<BlocksInGrid,threadsPerBlock>>>(salency,
                                           Imap,Omap,Cmap,
                                           rows*cols);
}

__global__ void matInfinityNorm(double *device_InMat,double *device_InfinityNorm,
                                int matRowSize, int matColSize, int threadDim){
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int tindex = (threadDim * tidx) + tidy;
    int maxNumThread = threadDim * threadDim; 
    int pass = 0;  
    int colCount, tCount ;
    int curRowInd;
    double tempInfinityNorm = 0.0;
    double rowMaxValue = 0.0;
      
    for( tCount = 1; tCount < maxNumThread; tCount++)
         device_InfinityNorm[tCount] = 0.0; 

    while( (curRowInd = (tindex + maxNumThread * pass))  < matRowSize ){
        rowMaxValue = 0.0;
        for( colCount = 0; colCount < matColSize; colCount++)
            rowMaxValue += abs(device_InMat[curRowInd* matRowSize + colCount]);
        tempInfinityNorm = ( tempInfinityNorm>rowMaxValue? tempInfinityNorm:rowMaxValue);
        pass++;
    }

    device_InfinityNorm[ tindex ] = tempInfinityNorm;
     __syncthreads();
   
    if(tindex == 0){
        for( tCount = 1; tCount < maxNumThread; tCount++)
            device_InfinityNorm[0] = device_InfinityNorm[0]> device_InfinityNorm[tCount]? device_InfinityNorm[0]: device_InfinityNorm[tCount]; 
    }
}


__global__ void meanMatrix(double *dMatrix, double *dMean, int dSize, int *d_mutex){
    __shared__ double cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    double temp = 0;
    while (tid < dSize) {
        temp += dMatrix[tid];
        tid  += blockDim.x * gridDim.x;
    }
    // set the cache values
    cache[cacheIndex] = temp;
    // synchronize threads in this block
    __syncthreads();

    int i = blockDim.x/2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }

    if(cacheIndex == 0){
		while(atomicCAS(d_mutex,0,1) != 0);  //lock
		*dMean += cache[0];
        atomicExch(d_mutex, 0);  //unlock
        
        *dMean = dMean[0]/dSize;
	}
}


__global__ void find_maximum(double *array, double *max, int dSize, int *d_mutex){
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	int stride = gridDim.x*blockDim.x;
	int offset = 0;

	__shared__ double cache[threadsPerBlock];

	double temp = -999999999.0;
	while(index + offset < dSize){
        temp = fmaxf(temp, array[index + offset]);
		offset += stride;
	}

	cache[threadIdx.x] = temp;

	__syncthreads();


	// reduction
	unsigned int i = blockDim.x/2;
	while(i != 0){
		if(threadIdx.x < i){
            cache[threadIdx.x] = fmax(cache[threadIdx.x], cache[threadIdx.x + i]);
		}

		__syncthreads();
		i /= 2;
	}

    if(threadIdx.x == 0){
		while(atomicCAS(d_mutex,0,1) != 0);  //lock
		*max = fmax(*max, cache[0]);
		atomicExch(d_mutex, 0);  //unlock
	}
}



__global__ void applyNormSum(double *dMap,double *dSupFeature, double *dMaxSupFeature, double *dMeanSupFeature,
                                          double *dInfFeature, double *dMaxInfFeature, double *dMeanInfFeature,
                                          int dSize){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    double SupCoeff = (dMaxSupFeature[0] - dMeanSupFeature[0])*(dMaxSupFeature[0] - dMeanSupFeature[0]);
    double InfCoeff = (dMaxInfFeature[0] - dMeanInfFeature[0])*(dMaxInfFeature[0] - dMeanInfFeature[0]);

    while (tid < dSize) {
        dMap[tid] += dSupFeature[tid]*SupCoeff + dInfFeature[tid]*InfCoeff;
        tid  += blockDim.x * gridDim.x;
    }
}

__global__ void absDifference(double *dDifference, double *dSup, double *dLow, int dSize){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < dSize) {
        double a = dSup[tid];
        double b = dLow[tid];
        dDifference[tid] = (a > b) ? (a - b) : (b - a);
        tid  += blockDim.x * gridDim.x;
    }
}

__global__ void sum3(double *d_result, 
                double *d_a, double *d_b, double *d_c, 
                int dSize){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < dSize) {
        d_result[tid] = d_a[tid] + d_b[tid] +d_c[tid];
        tid  += blockDim.x * gridDim.x;
    }
}

__global__ void divScalarMatrix(double *dMatrix, double *dScalar, int dSize){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < dSize) {
        dMatrix[tid] = dMatrix[tid]/dScalar[0];
        tid  += blockDim.x * gridDim.x;
    }
}
